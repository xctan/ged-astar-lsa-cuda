#include "hip/hip_runtime.h"

#include <assert.h>
#include <stdio.h>
#include "list.h"

list **lists_create(int lists, int capacity) {
    list **lists_cpu = (list**)malloc(lists * sizeof(list*));
    list **lists_gpu = NULL;
    for (int i = 0; i < lists; i++) {
        lists_cpu[i] = list_create(capacity);
    }
    HANDLE_RESULT(hipMalloc(&lists_gpu, lists * sizeof(list*)));
    HANDLE_RESULT(hipMemcpy(lists_gpu, lists_cpu, lists * sizeof(list*), hipMemcpyDefault));
    free(lists_cpu);
    return lists_gpu;
}

list *list_create(int capacity) {
    list list_cpu;
    list *list_gpu;
    list_cpu.length = 0;
    list_cpu.capacity = capacity;
    HANDLE_RESULT(hipMalloc(&(list_cpu.arr), (capacity + 1) * sizeof(state*)));
    HANDLE_RESULT(hipMalloc(&list_gpu, sizeof(struct list)));
    HANDLE_RESULT(hipMemcpy(list_gpu, &list_cpu, sizeof(struct list),
                hipMemcpyDefault));
    return list_gpu;
}

void lists_destroy(list **lists_gpu, int lists) {
    list **lists_cpu = (list**)malloc(lists * sizeof(list*));
    HANDLE_RESULT(hipMemcpy(lists_cpu, lists_gpu, lists * sizeof(list*), hipMemcpyDefault));
    for (int i = 0; i < lists; i++) {
        list_destroy(lists_cpu[i]);
    }
    HANDLE_RESULT(hipFree(lists_gpu));
    free(lists_cpu);
}

void list_destroy(list *list_gpu) {
    list list_cpu;
    HANDLE_RESULT(hipMemcpy(&list_cpu, list_gpu, sizeof(struct list),
                hipMemcpyDefault));
    HANDLE_RESULT(hipFree(list_cpu.arr));
    HANDLE_RESULT(hipFree(list_gpu));
}
__device__ void list_clear(list *list) {
    list->length = 0;
}

__device__ void list_insert(list *list, state *state) {
    int index = atomicAdd(&(list->length), 1);
    assert(index < list->capacity);
    list->arr[index] = state;
}

__device__ void list_remove(list *list, int index) {
    assert(list->length < list->capacity);
    list->arr[index] = NULL;
}

__device__ state *list_get(list *list, int index) {
    assert(index < list->length);
    return list->arr[index];
}